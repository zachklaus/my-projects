#include "hip/hip_runtime.h"

#include "matmultKernel.h"

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){


  float *Asub, *Bsub, *Csub;

  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * block_col];

  float Cvalue0 = 0;
  float Cvalue1 = 0;
  float Cvalue2 = 0;
  float Cvalue3 = 0;

  for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){

    Asub = &A.elements[A.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * m];
    Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * block_col];

    __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
    __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];

    shared_A[thread_row][thread_col] = Asub[thread_row * A.stride + thread_col];
    shared_A[thread_row + blockDim.y][thread_col + blockDim.x] = Asub[(thread_row + blockDim.y )* A.stride + (thread_col + blockDim.x)];
    shared_A[thread_row + 2 * blockDim.y][thread_col + 2 * blockDim.x] = Asub[(thread_row + 2 * blockDim.y )* A.stride + (thread_col + 2 * blockDim.x)];
    shared_A[thread_row + 3 * blockDim.y][thread_col + 3 * blockDim.x] = Asub[(thread_row + 3 * blockDim.y )* A.stride + (thread_col + 3 * blockDim.x)];

    shared_B[thread_row][thread_col] = Bsub[thread_row * B.stride + thread_col];
    shared_B[thread_row + blockDim.y][thread_col + blockDim.x] = Bsub[(thread_row + blockDim.y )* B.stride + (thread_col + blockDim.x)];
    shared_B[thread_row + 2 * blockDim.y][thread_col + 2 * blockDim.x] = Bsub[(thread_row + 2 * blockDim.y )* B.stride + (thread_col + 2 * blockDim.x)];
    shared_B[thread_row + 3 * blockDim.y][thread_col + 3 * blockDim.x] = Bsub[(thread_row + 3 * blockDim.y )* B.stride + (thread_col + 3 * blockDim.x)];

    __syncthreads();

#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; ++e)
       Cvalue0 += shared_A[thread_row][e] * shared_B[e][thread_col];

#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; ++e)
       Cvalue1 += shared_A[thread_row + blockDim.y][e] * shared_B[e][thread_col + blockDim.x];

#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; ++e)
      Cvalue2 += shared_A[thread_row + 2 * blockDim.y][e] * shared_B[e][thread_col + 2 * blockDim.x];

#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; ++e)
      Cvalue3 += shared_A[thread_row + 3 * blockDim.y][e] * shared_B[e][thread_col + 3 * blockDim.x];

    __syncthreads();
  }

  Csub[thread_row * C.stride + thread_col] = Cvalue0;
  Csub[(thread_row + blockDim.y) * C.stride + (thread_col + blockDim.x)] = Cvalue1;
  Csub[(thread_row + 2 * blockDim.y) * C.stride + (thread_col + 2 * blockDim.x)] = Cvalue2;
  Csub[(thread_row + 3 * blockDim.y) * C.stride + (thread_col + 3 * blockDim.x)] = Cvalue3;
}
