//Author: Zachary Klausner
//Class: CS475
//Assignment: PA5

// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <limits.h>
#include <float.h>
#include <time.h>
#include <sys/time.h>
#include <sys/errno.h>
#include <omp.h>

#define max(x, y)   ((x)>(y) ? (x) : (y))
#define min(x, y)   ((x)>(y) ? (y) : (x))

__device__ void multiplyMatrices(float*, float*, float*, long);
void print_matrix_array(float*, long, long, char*);

void print_matrix_array(float* matrix, long N_or_G, long B, char* name) {
  long end = N_or_G*B*B;
  for (int i = 0; i < end; i++) {
    printf("%s[%d] = %f\n", name, i, matrix[i]);
  }
}

void MMScan(float ***X, float ***Y, long start, long end, long size){
  long n, i, j, k;
  for(i=0; i <= size-1; i+=1)
    {
      for(j=0; j <= size-1; j+=1)
	{
	  Y[start][i][j] = X[start][i][j];
	}
    }

#ifdef FAUX  // incorrect parallelization
#pragma omp parallel for
#endif // incorrect parallelization
  for(n=start+1; n <= end; n+=1)
    {
      for(i=0; i < size; i+=1)
	{
	  for(j=0; j < size; j+=1)
	    {
	      float acc = 0;
	      for(k=0; k<size; k++){
		acc = acc + Y[n-1][i][k] * X[n][k][j];
	      }
	      Y[n][i][j] = acc;
	    }
	}
    }
}

__global__ void Phase_1(float* X, float* R1,long N, long B) {

  float* T1 = (float*)malloc(sizeof(float)*((B) * (B)));
  float* T2 = (float*)malloc(sizeof(float)*((B) * (B)));
  float* T3 = (float*)malloc(sizeof(float)*((B) * (B)));

  float* identity = (float*)malloc(sizeof(float)*((B) * (B)));

  long G = gridDim.x;
  long n = N/G;
  long start = n * blockIdx.x * (B*B);
  long end = start + (G*B*B);

  int T1_index = 0;
  int T2_index = 0;
  int T3_index = 0;

  int row_loc = 0;
  int col_loc = 0;

  for (int i = 0; i < B*B; i++) {
    if (row_loc == col_loc) {
      identity[i] = 1.0;
    }
    if ((i+1) % B == 0 && i != 0) {
      row_loc++;
      col_loc = 0.0;
      continue;
    }
    col_loc++;
  }

  int identity_index = 0;
  for (int i = blockIdx.x*B*B; i < blockIdx.x*B*B + B*B; i++) {
    R1[i] = identity[identity_index];
    identity_index++;
  }

  for (int i = 0; i < n; i++) {

    T1_index = 0;
    for (int j = start + (i*B*B); j < (start + (i*B*B)) + B*B; j++) {
      T1[T1_index] = X[j];
      T1_index++;
    }

    T2_index = 0;
    for (int j = blockIdx.x*B*B; j < (blockIdx.x*B*B) + B*B; j++) {
      T2[T2_index] = R1[j];
      T2_index++;
    }

    multiplyMatrices(T1,T2,T3,B);

    __syncthreads();

    T3_index = 0;
    for (int j = blockIdx.x*B*B; j < (blockIdx.x*B*B) + B*B; j++) {
      R1[j] = T3[T3_index];
      T3_index++;
    }
  }
}

__global__ void Phase_2(float* R1, float* R2, long N, long B, long G) {

  float* T1 = (float*)malloc(sizeof(float)*((B) * (B)));
  float* T2 = (float*)malloc(sizeof(float)*((B) * (B)));
  float* T3 = (float*)malloc(sizeof(float)*((B) * (B)));

  int T1_index = 0;
  int T2_index = 0;
  int T3_index = 0;

  for (int i = 1; i < G; i++) {

    T1_index = 0;
    for (int j = (i-1)*B*B; j < ((i-1)*B*B) + B*B; j++) {
      T1[T1_index] = R2[j];
      T1_index++;
    }

    T2_index = 0;
    for (int j = i*B*B; j < (i*B*B) + B*B; j++) {
      T2[T2_index] = R1[j];
      T2_index++;
    }

    multiplyMatrices(T1,T2,T3,B);

    __syncthreads();

    T3_index = 0;
    for (int j = i*B*B; j < (i*B*B) + B*B; j++) {
      R2[j] = T3[T3_index];
      T3_index++;
    }
  }
}

__global__ void Phase_3(float* R2, float* Y, long N, long B) {

  float* T1 = (float*)malloc(sizeof(float)*((B) * (B)));
  float* T2 = (float*)malloc(sizeof(float)*((B) * (B)));
  float* T3 = (float*)malloc(sizeof(float)*((B) * (B)));

  int T1_index = 0;
  int T2_index = 0;
  int T3_index = 0;

  long G = gridDim.x;
  long n = N/G;
  long start = n * blockIdx.x * (B*B);
  long end = start + (G*B*B);

  if (blockIdx.x != 0) {
    T1_index = 0;
    for (int i = (blockIdx.x-1)*B*B; i < ((blockIdx.x-1)*B*B) + B*B; i++) {
      T1[T1_index] = R2[i];
      T1_index++;
    }
  }
  else {
    int row_loc = 0;
    int col_loc = 0;

    for (int i = 0; i < B*B; i++) {
      if (row_loc == col_loc) {
        T1[i] = 1.0;
      }
      if ((i+1) % B == 0 && i != 0) {
        row_loc++;
        col_loc = 0.0;
        continue;
      }
      col_loc++;
    }

    for (int i = 0; i < N; i ++) {
      T2_index = 0;
      for (int j = (i*B*B) + blockIdx.x*B*B; j < ((i*B*B)+blockIdx.x*B*B) + B*B; j++) {
        T2[T2_index] = Y[j];
        T2_index++;
      }

      multiplyMatrices(T1,T2,T3,B);

      __syncthreads();

      T3_index = 0;
      for (int j = (i*B*B) + blockIdx.x*B*B; j < ((i*B*B)+blockIdx.x*B*B) + B*B; j++) {
        Y[j] = T3[T3_index];
        T3_index++;
      }

      for (int j = 0; j < B*B; j++) {
        T1[j] = T3[j];
      }

    }
  }
}

__device__ void multiplyMatrices(float* matA, float* matB, float* matC, long width) {

  for (int i = 0; i < width; i++) {
    for (int j = 0; j < width; j++) {
        float sum = 0.0;
        for (int k = 0; k < width; k++)
            sum = sum + matA[i * width + k] * matB[k * width + j];
            matC[i * width + j] = sum;
          }
  }

}
