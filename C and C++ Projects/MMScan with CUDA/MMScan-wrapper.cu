//Author: Zachary Klausner
//Class: CS475
//Assignment: PA5

// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <limits.h>
#include <float.h>
#include <time.h>
#include <sys/time.h>
#include <sys/errno.h>
#include <omp.h>

#define max(x, y)   ((x)>(y) ? (x) : (y))
#define min(x, y)   ((x)>(y) ? (y) : (x))

// Common Macros
#define mallocCheck(v,s,d) if ((v) == NULL) { printf("Failed to allocate memory for %s : size=%lu\n", "sizeof(d)*(s)", sizeof(d)*(s)); exit(-1); }
#define EPSILON 1.0E-6
#define G 1000
#define S 16

__global__ void Phase_1(float*, float*, long, long);
__global__ void Phase_2(float*, float*, long, long, long);
__global__ void Phase_3(float*, float*, long, long);

__device__ void multiplyMatrices(float*, float*, float*, int);

void print_matrix_array(float*, long, long, char*);
void MMScan(float***, float***, long, long, long);

//main
int main(int argc, char** argv) {
  //Check number of args
  if (argc <= 2) {
    printf("Number of argument is smaller than expected.\n");
    printf("Expecting N,B\n");
    exit(0);
  }

  if (atoi(argv[1]) % G != 0) {
    printf("N is not divisible by G = %d!\n", G);
    exit(0);
  }

  //char *end = 0;
  char *val = 0;
  //Read Parameters
  //Initialization of N
  val = argv[1];
  long N = atoi(val);

  //Initialization of B
  val = argv[2];
  long B = atoi(val);

  long tuning = 0;
  //Additional args?
  if(argc > 3)
    {
      val = argv[3];
      tuning = atoi(val);
    }

  ///Parameter checking
  if (!((N >= 1 && B >= 1))) {
    printf("The value of parameters are not valid.\n");
    exit(-1);
  }

  //CPU Memory Allocation
  long n, i, j; //k;

  float* _lin_X_seq = (float*)malloc(sizeof(float)*((N) * (B) * (B)));
  mallocCheck(_lin_X_seq, ((N) * (B) * (B)), float);
  float*** X_seq = (float***)malloc(sizeof(float**)*(N));
  mallocCheck(X_seq, (N), float**);
  for (n=0;n < N; n++) {
    X_seq[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(X_seq[n], (B), float*);
    for (i=0;i < B; i++) {
      X_seq[n][i] = &_lin_X_seq[(n*((B) * (B))) + (i*(B))];
    }
  }

  float* _lin_Y_seq = (float*)malloc(sizeof(float)*((N) * (B) * (B)));
  mallocCheck(_lin_Y_seq, ((N) * (B) * (B)), float);
  float*** Y_seq = (float***)malloc(sizeof(float**)*(N));
  mallocCheck(Y_seq, (N), float**);
  for (n=0;n < N; n++) {
    Y_seq[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(Y_seq[n], (B), float*);
    for (i=0;i < B; i++) {
      Y_seq[n][i] = &_lin_Y_seq[(n*((B) * (B))) + (i*(B))];
    }
  }

  float* _lin_X = (float*)malloc(sizeof(float)*((N) * (B) * (B)));
  mallocCheck(_lin_X, ((N) * (B) * (B)), float);
  float*** X = (float***)malloc(sizeof(float**)*(N));
  mallocCheck(X, (N), float**);
  for (n=0;n < N; n++) {
    X[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(X[n], (B), float*);
    for (i=0;i < B; i++) {
      X[n][i] = &_lin_X[(n*((B) * (B))) + (i*(B))];
    }
  }

  float* _lin_Y = (float*)malloc(sizeof(float)*((N) * (B) * (B)));
  mallocCheck(_lin_Y, ((N) * (B) * (B)), float);
  float*** Y = (float***)malloc(sizeof(float**)*(N));
  mallocCheck(Y, (N), float**);
  for (n=0;n < N; n++) {
    Y[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(Y[n], (B), float*);
    for (i=0;i < B; i++) {
      Y[n][i] = &_lin_Y[(n*((B) * (B))) + (i*(B))];
    }
  }

  //R1 and R2 allocations
  float* _lin_R1 = (float*)malloc(sizeof(float)*((G) * (B) * (B)));
  mallocCheck(_lin_R1, ((G) * (B) * (B)), float);
  float*** R1 = (float***)malloc(sizeof(float**)*(G));
  mallocCheck(R1, (G), float**);
  for (n=0;n < G; n++) {
    R1[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(R1[n], (B), float*);
    for (i=0;i < B; i++) {
      R1[n][i] = &_lin_R1[(n*((B) * (B))) + (i*(B))];
    }
  }

  float* _lin_R2 = (float*)malloc(sizeof(float)*((G) * (B) * (B)));
  mallocCheck(_lin_R2, ((G) * (B) * (B)), float);
  float*** R2 = (float***)malloc(sizeof(float**)*(G));
  mallocCheck(R2, (G), float**);
  for (n=0;n < G; n++) {
    R2[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(R2[n], (B), float*);
    for (i=0;i < B; i++) {
      R2[n][i] = &_lin_R2[(n*((B) * (B))) + (i*(B))];
    }
  }

  float* _lin_Temp = (float*)malloc(sizeof(float)*((N) * (B) * (B)));
  mallocCheck(_lin_Temp, ((N) * (B) * (B)), float);
  float*** Temp = (float***)malloc(sizeof(float**)*(N));
  mallocCheck(Temp, (N), float**);
  for (n=0;n < N; n++) {
    Temp[n] = (float**)malloc(sizeof(float*)*(B));
    mallocCheck(Temp[n], (B), float*);
    for (i=0;i < B; i++) {
      Temp[n][i] = &_lin_Temp[(n*((B) * (B))) + (i*(B))];
    }
  }

  //Initialization of rand
  srand((unsigned)time(NULL));

  //Input Initialization

#if defined (RANDOM)
  float x, y; //tmp;
  x = (float) rand();
  for(n=0; n <= N-1; n+=1) {
    y = (float) rand();
    for(i=0; i <= B-1; i+=1)
	   for(j=0; j <= B-1; j+=1) {
	      X[n][i][j] = y/(B*x);
        X_seq[n][i][j] = y/(B*x);
      }
      x = y;
    }
#else  // not random
  for(i=0; i <= B-1; i+=1)
    for(j=0; j <= B-1; j+=1) {
      X[0][i][j] = (float) 1.0;
      X_seq[0][i][j] = (float) 1.0;
    }  // all 1s
  for(n=1; n <= N-1; n+=1)
  {
      for(i=0; i <= B-1; i+=1)
	{
	  for(j=0; j <= B-1; j+=1)
	    {
#if defined (INTERACTIVE)
	      {
          float temp;
		      printf("X[%ld][%ld][%ld]= ", n, i, j);
		      scanf("%f", &temp);
          X[n][i][j] = temp;
          X_seq[n][i][j] = temp;
	      }
#else // neither random not interactive, i.e., default
{
	      X[n][i][j] = (float) (n+1)/((float) (B*n));
        X_seq[n][i][j] = (float) (n+1)/((float) (B*n));
}
#endif
	    }
	}
  }

#endif

  //Timing
  struct timeval time;
  double elapsed_time1, elapsed_time2;

  //Call the main computation

  //**************************************************************************//
  /*                     START OF THE SCAN COMPUTATION                        */
  //**************************************************************************//
  /* int p = omp_get_num_procs(); */
  /* printf("There are %ld threads\n", p); */

  gettimeofday(&time, NULL);
  elapsed_time1 = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000);

  hipSetDevice(0);

  // printf("X:\n");
  // print_matrix_array(_lin_X, N, B, "X");

  // printf("\nBEFORE:\n");
  // print_matrix_array(_lin_R1, G, B, "R1");
  //R1[0][0][0] = 5.5;

  float* X_GPU;
  float* Y_GPU;
  float* R1_GPU;
  float* R2_GPU;

  hipMalloc(&X_GPU, (sizeof(float)*(N*B*B)));
  hipMalloc(&Y_GPU, (sizeof(float)*(N*B*B)));
  hipMalloc(&R1_GPU, (sizeof(float)*(G*B*B)));
  hipMalloc(&R2_GPU, (sizeof(float)*(G*B*B)));

  hipMemcpy(X_GPU, _lin_X, (sizeof(float)*(N*B*B)), hipMemcpyHostToDevice);
  hipMemcpy(Y_GPU, _lin_X, (sizeof(float)*(N*B*B)), hipMemcpyHostToDevice);
  hipMemcpy(R1_GPU, _lin_R1, (sizeof(float)*(G*B*B)), hipMemcpyHostToDevice);
  hipMemcpy(R2_GPU, _lin_R2, (sizeof(float)*(G*B*B)), hipMemcpyHostToDevice);

  // printf("\nBEFORE:\n");
  // print_matrix_array(_lin_R1, G, B, "R1");

  Phase_1<<<G,S>>>(X_GPU, R1_GPU, N, B);

  hipMemcpy(_lin_R1, R1_GPU, (sizeof(float)*(G*B*B)), hipMemcpyDeviceToHost);

  // printf("\nR1 AFTER:\n");
  // print_matrix_array(_lin_R1, G, B, "R1");

  Phase_2<<<1,S>>>(R1_GPU, R2_GPU, N, B, G);

  hipMemcpy(_lin_R2, R2_GPU, (sizeof(float)*(G*B*B)), hipMemcpyDeviceToHost);

  // printf("\nR2 AFTER:\n");
  // print_matrix_array(_lin_R2, G, B, "R2");

  Phase_3<<<G,S>>>(R2_GPU, Y_GPU, N, B);

  hipMemcpy(_lin_Y, Y_GPU, (sizeof(float)*(N*B*B)), hipMemcpyDeviceToHost);

  // printf("\nY AFTER:\n");
  // print_matrix_array(_lin_Y, N, B, "Y");

  gettimeofday(&time, NULL);
  elapsed_time1 = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000) - elapsed_time1;

  float ***tmp_ptr = Temp;
  Temp = Y;  Y = tmp_ptr; // swap Temp and Y so that the next call computes Y
			  // with the standard sequential algorithm

  gettimeofday(&time, NULL);
  elapsed_time2 = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000);

  // the provided seqential algorithm

  MMScan(X_seq, Y_seq, 0, N-1, B);

  gettimeofday(&time, NULL);
  elapsed_time2 = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000) - elapsed_time2;

  int xDirection,yDirection,zDirection;

  for (int i = 0; i < N*B*B; i++) {
    zDirection = i % B;
    yDirection = (i / B) % B;
    xDirection = i / (B * B);

    Y[xDirection][yDirection][zDirection] = _lin_Y[i];

  }

  //**************************************************************************//
  /*                       END OF THE SCAN COMPUTATION                        */
  /*                                                                          */
  /*                    PRINT OUTPUTS (DEPENDING ON FLAGS)                    */
  //**************************************************************************//

#ifdef INTERACTIVE
  //Print Outputs Interactively

  for(n=0; n <= N-1; n+=1)
    {
      printf("Y[%ld][i][j]= \n", n);
      for(i=0; i <= B-1; i+=1)
	{
	  for(j=0; j <= B-1; j+=1)
	    {
	      printf("%10g ", Y[n][i][j]);
	    }
	  printf("\n");
	}
      printf("\n");
    }
#endif

#ifdef VERBOSE
  //Print Inputs and Outputs (leading and trailing, no more than 5 each)

  //  First print the first five (X, Y)
  for(n=0; n <= min(N-1, 5); n+=1)
    {
      printf("\tX[%ld][i][j], \tY[%ld][i][j], \n", n, n);
      for(i=0; i <= B-1; i+=1)
	{
	  for(j=0; j <= B-1; j+=1)
	    {
	      printf("%10g ", X[n][i][j]);
	    }
	  printf("\t");
	  for(j=0; j <= B-1; j+=1)
	    {
	      printf("%10g ", Y[n][i][j]);
	    }
	  printf("\n");
	}
      printf("\n");
    }

  //  Then print the last five (X, Y)
  for(n=max(5, N-5); n <= N-1; n+=1)
    {
      printf("\tX[%ld][i][j], \tY[%ld][i][j], \n", n, n);
      for(i=0; i <= B-1; i+=1)
	{
	  for(j=0; j <= B-1; j+=1)
	    {
	      printf("%10g ", X[n][i][j]);
	    }
	  printf("\t");
	  for(j=0; j <= B-1; j+=1)
	    {
	      printf("%10g ", Y[n][i][j]);
	    }
	  printf("\n");
	}
      printf("\n");
    }
#endif

#if defined CHECKING
  // Compare the values in Y (CUDA result) and Y_seq (sequential result)
  long error_count = 0;

  for(n=0; n <= N-1; n+=1)
    {
      for(i=0; i <= B-1; i+=1)
	{
	  for(j=0; j <= B-1; j+=1)
	    {
	      if (fabs(Y[n][i][j]-Y_seq[n][i][j]) > EPSILON)
		{error_count += 1;
		  printf ("Y[%ldl][%ldl][%ldl] = %f, \tY_seq[%ldl][%ldl][%ldl] = %f\n",
			  n, i, j, Y[n][i][j], n, i, j, Y_seq[n][i][j]);
		}
	    }
	}
    }
  printf("The total number of errors is %ld\n", error_count);
#endif

  // timing information
  printf("Execution time for DNC:\t%lf sec.\n", elapsed_time1);
  printf("Execution time for SEQ:\t%lf sec.\n", elapsed_time2);

  //Memory Free
   free(_lin_X);
   for (n=0;n < N; n++) {
     free(X[n]);
   }
   free(X);

   free(_lin_Y);
   for (n=0;n < N; n++) {
     free(Y[n]);
  }
   free(Y);

   free(_lin_R1);
   for (n=0;n < G; n++) {
     free(R1[n]);
  }
   free(R1);

   free(_lin_R2);
   for (n=0;n < G; n++) {
     free(R2[n]);
  }
   free(R2);

   free(_lin_Temp);
   for (n=0;n < N; n++) {
     free(Temp[n]);
  }
   free(Temp);

   hipFree(R1_GPU);
   hipFree(R2_GPU);
   hipFree(X_GPU);
   hipFree(Y_GPU);

  return EXIT_SUCCESS;
}


//Common Macro undefs
#undef EPSILON
