
#include <hip/hip_runtime.h>
///
/// vecMaxKernel00.cu : code computes partiol answer per each thread.
/// By Waruna Ranasinghe
/// Created: 15 Aug 2017
/// Last Modified:

/// The code computes partial answer for reduction over max per a thread.
/// Each thread computes max of a consective chunck of data of size C.
/// The memory loads are not coalesced.


/*
 * A - input vector of floats of size G*B*C
 * reductions - output of partial answers compted by each thread
 * C - chunck size - number of elements processed by ech thread
 */
__global__ void reduce(const float* A, float* reductions, int C) {

  	int tid = threadIdx.x; 		//Thread index within a thread block
  	int blockid = blockIdx.x; //Block index within the grid
  	int B = blockDim.x; 			//numer of threads per block

  	//The index of the array corresponds to the start of a thread block
  	int start_of_the_block = blockid*B*C;
    int end_of_the_block = start_of_the_block + B*C;
  	reductions[blockid*B + tid] = 0.0f;

  	for (int i=start_of_the_block + tid; i < end_of_the_block; i += B) {
  		reductions[blockid*B+tid] = max(reductions[blockid*B+tid],A[i]);
  	}
}
