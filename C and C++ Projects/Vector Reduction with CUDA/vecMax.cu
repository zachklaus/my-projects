#include "hip/hip_runtime.h"
///
/// vecMax.cu : contains the host code for the assignment:
/// vector reduction with max operation.
/// By Waruna Ranasinghe 
/// Created: 15 Aug 2017
/// Last Modified: 

///NOTE: Please read all the comments before modifying the file. Failure to maintain
//the required variable names and function names will be penalized. 

/// The length of the vector is G*B*C where G is the number of thread blocks
//in the grid, B is the number of threds in the thread block and C is the
//chucnk size computed by one thread

//Includes
#include <stdio.h>
#include "vecMaxKernel.h"
#include "timer.h"

//defines

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                      \
  hipError_t _m_cudaStat = value;                    \
  if (_m_cudaStat != hipSuccess) {                   \
    fprintf(stderr, "Error: %s at line %d in file %s\n",          \
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
    exit(1);                              \
  } }

#define epsilon (float)1e-4
//NOTE: DO NOT define verbose option in your submission

/// main
int main(int argc, char** argv) {
	int G; //number of thread blocks in the grid	
	int B; //number of threads per thread block
	int C; //number of elements processed by one thread (chunk size)A
	int n; //size of vector G*B*C

	//host variables
	float result; // your final answer should be assigned to this variable
	float *h_A; //input vector of floats
	float *h_reductions; //output from device - partial results from gpu. G*B
											 //number of partial results one result per thread
	//device variables
	float *d_A; // input vector of floats
	float *d_reductions; //output from device - partial results from gpu. G*B
											 //number of partial results one result per thread
	
	if (argc != 4) {
		printf("Usage: %s G B n\n\tG - number of thread blocks in the grid\n\tB -\
 number of threads per thread block\n\tn - size of the vector\n", argv[0]);
		exit(0);
	} 

	//init size params
	G = atoi(argv[1]);
	B = atoi(argv[2]);
	n = atoi(argv[3]);

	if (n % (G*B) != 0) {
		printf("Vector length (n=%d) is not completely divisible by (G*B=%d).\n", n, G*B);
		exit(0);
	}

	//computing the size of the vector
	C = n/(G*B);	

	//allocating memory for host variables
	h_A = (float *)malloc(n*sizeof(float));
	if (h_A==NULL) {
		fprintf(stderr, "Failed to allocate host vector h_A!\n"); 
		exit(-1);
	}

	h_reductions = (float *)malloc(G*B*sizeof(float));
	if (h_reductions==NULL) {
		fprintf(stderr, "Failed to allocate host vector h_reductions!\n");
		exit(-1);
	}

	//initializing input vector
	for (int i=0; i<n; i++) {
		h_A[i]=(n-i)/1.0f;
	}	

	CUDA_CHECK_RETURN(hipSetDevice(0));

	//Allocating the device memory for input vector. 
	//CUDA_CHECK_RETURN macro verifies the return code. If it refers to an error
	//code, it prints the error message and exit the program
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_A, n*sizeof(float)));	

	//allocate device memory for the partial output
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_reductions, G*B*sizeof(float)));	

	initialize_timer ();
	start_timer();
	//copy the host input vector h_A to the device input vector d_A in device
	//memory	
	CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, n*sizeof(float), hipMemcpyHostToDevice));

	double time_input, time_compute, time_output; 
  /* Start Timer */
	//initialize_timer ();
	//start_timer();

	stop_timer();
	time_input=elapsed_time();
	reset_timer();
	start_timer();
	//Launch the reduce CUDA kernel with G blocks per grid and B threads per
	//block
	reduce<<<G, B>>>(d_A, d_reductions, C);

	//check whether there were errors while launching the CUDA kernel
	CUDA_CHECK_RETURN(hipGetLastError());

	//wait for the kernel to finish
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	/* stop timer */
	stop_timer();
	time_compute=elapsed_time ();
	reset_timer();
	start_timer();

	//Copy partial results back to host
	CUDA_CHECK_RETURN(hipMemcpy(h_reductions, d_reductions, G*B*sizeof(float), hipMemcpyDeviceToHost));

	//assuming input values are >= 0
	result = 0.0f;
	for (int i=0; i<G*B; i++) {
		result = max(result,h_reductions[i]);
	}

	stop_timer();
	time_output=elapsed_time ();
	printf("Result: %f\n Time to copy input: %f\n Compute time: %f\n Time to  copy output and generate final answer: %f\n",  result, time_input, time_compute, time_output);

	//cleaning up
	free(h_A);
	free(h_reductions);

	return 0;
}

